#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "gif.h"
#include <vector>
#include <cstdint>
#include <stdio.h>


//Values
#define X 1000
#define Y 2000
#define T 5
#define IT 1000
#define output "test.gif"

//Device variables
__device__ bool dev_field[X][Y];

__device__ bool dev_newField[X][Y];

__device__ int dev_neighbours[X][Y];

__device__ uint8_t dev_image[X * Y * 4];

__device__ int dev_result;


//Host variables
int result = 0;

bool hst_field[X][Y];
uint8_t hst_image[X * Y * 4];



/*
	Reset NeighbourTable with zeros
	And the living cell counter
*/
__global__ void ResetNeighbourTable()
{
	
	if (blockIdx.x == 0 && blockIdx.y == 0)
	{
		dev_result = 0;
	}
	dev_neighbours[blockIdx.x][blockIdx.y] = 0;

	
	
}

/*
	Neighbour calculation
*/


__global__ void CalculateCellNeighbours()
{
	
	__shared__ bool shr_neighbours[3][3];

	
	__shared__ uint8_t minX, maxX, minY, maxY;

	minX = 0;
	maxX = 2;
	minY = 0;
	maxY = 2;


	//SETUP
	if (threadIdx.x == 0 && threadIdx.y == 0)
	{

		if (blockIdx.x == 0)
		{
			shr_neighbours[0][0] = 0;
			shr_neighbours[0][1] = 0;
			shr_neighbours[0][2] = 0;
			
			minX = 1;
		
		}
		if (blockIdx.y == 0)
		{
			shr_neighbours[0][0] = 0;
			shr_neighbours[1][0] = 0;
			shr_neighbours[2][0] = 0;
			
			minY = 1;
		}
		if (blockIdx.x == X)
		{
			shr_neighbours[0][2] = 0;
			shr_neighbours[1][2] = 0;
			shr_neighbours[2][2] = 0;

			maxX = 1;
			
		}
		if (blockIdx.y == Y)
		{
			
			shr_neighbours[2][0] = 0;
			shr_neighbours[2][1] = 0;
			shr_neighbours[2][2] = 0;
			maxY = 1;
		}



	}

	__syncthreads();
	//LOADING
	if (threadIdx.x >= minX && threadIdx.x <= maxX && threadIdx.y >= minY && threadIdx.y <= maxY)
	{
		shr_neighbours[threadIdx.x][threadIdx.y] = dev_field[blockIdx.x - 1 + threadIdx.x][blockIdx.y - 1 + threadIdx.y];
			
	}
	//__syncthreads();

	//NO SELFREPORT
	if (threadIdx.x == 1 && threadIdx.y == 1)
	{
		shr_neighbours[1][1] = 0;
	}
	
	
	

	//SUM NEIGHBOURS(bool true = int 1)
	atomicAdd(&dev_neighbours[blockIdx.x][blockIdx.y], shr_neighbours[threadIdx.x][threadIdx.y]);

	

	

}
/*
	After Calculating the Neighbours table, then using it by the rules.
*/
__global__ void SetNewField()
{
	__shared__ bool shr_alive; 
	__shared__ int shr_neighbours;
	
	
	
	shr_alive = dev_field[blockIdx.x][blockIdx.y];
	shr_neighbours = dev_neighbours[blockIdx.x][blockIdx.y];

	

	
	//Dying condition
	if (shr_alive && (shr_neighbours < 2 || shr_neighbours > 3))
	{
		dev_newField[blockIdx.x][blockIdx.y] = false;
	}
	//Revive condition
	else if (!shr_alive && shr_neighbours == 3)
	{
		dev_newField[blockIdx.x][blockIdx.y] = true;
	}
	//Otherwise just copy
	else
	{
		//dev_newField[blockIdx.x][blockIdx.y] = dev_field[blockIdx.x][blockIdx.y];
		dev_newField[blockIdx.x][blockIdx.y] = shr_alive;
	}
	//Counting the living cells
	//atomicAdd(&dev_result, dev_field[blockIdx.x][blockIdx.y]);
	atomicAdd(&dev_result, shr_alive);
	
}

/*
	Copy Method
*/
__global__ void CopyNewToOld()
{
	dev_field[blockIdx.x][blockIdx.y] = dev_newField[blockIdx.x][blockIdx.y];
}
/*
	Copy Convert Method
*/
__global__ void MakeImage()
{
	dev_image[4 * Y * blockIdx.x + blockIdx.y * 4 + 0] = dev_newField[blockIdx.x][blockIdx.y] * 255;
	dev_image[4 * Y * blockIdx.x + blockIdx.y * 4 + 1] = dev_newField[blockIdx.x][blockIdx.y] * 255;
	dev_image[4 * Y * blockIdx.x + blockIdx.y * 4 + 2] = dev_newField[blockIdx.x][blockIdx.y] * 255;
	dev_image[4 * Y * blockIdx.x + blockIdx.y * 4 + 3] = dev_newField[blockIdx.x][blockIdx.y] * 255;
}


int main()
{
	int width = Y;
	int height = X;

	int delay = T;
	
	auto filename = output;
	
	

	GifWriter g;

	GifBegin(&g, filename, width, height, delay);

	hst_field[0][0] = true;
	hst_field[0][1] = true;
	hst_field[1][0] = true;

	//spin
	hst_field[5][5] = true;
	hst_field[5][6] = true;
	hst_field[5][7] = true;

	//go
	hst_field[10][10] = true;
	hst_field[11][11] = true;
	hst_field[11][12] = true;
	hst_field[12][10] = true;
	hst_field[12][11] = true;


	//starting field copy
	hipMemcpyToSymbol(HIP_SYMBOL(dev_field), hst_field, X * Y * sizeof(bool));
	
	
	for (size_t i = 0; i < IT; i++)
	{
		ResetNeighbourTable << <dim3(X,Y), 1 >> > ();

		CalculateCellNeighbours << <dim3(X,Y), dim3(3, 3) >> > ();

		SetNewField << <dim3(X,Y), 1 >> > ();

		MakeImage << <dim3(X, Y), 1 >> > ();
		
		CopyNewToOld << < dim3(X, Y), 1 >> > ();
		
		hipMemcpyFromSymbol(&result, HIP_SYMBOL(dev_result), sizeof(int));

		hipMemcpyFromSymbol(hst_image, HIP_SYMBOL(dev_image), X * Y * 4 * sizeof(uint8_t));
		GifWriteFrame(&g, hst_image, width, height, delay);

		//If there are 2 living cell, then end the simulation.(Because in the next iteration, all of them will die)
		if (result < 2)
		{
			break;
		}
		
	}


	

	

	GifEnd(&g);

	return 0;
}